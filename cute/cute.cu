#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <memory.h>
#define w 640
#define h 480
#define ysize w*h
#define blks 8
#define thnum 640
#define cysize blks*thnum //30720
#define cuvsize cysize/4 //7680
#define cui h/blks
//======namespace=======//
using namespace std;
using namespace cv;
//======================//
//===========kernel function=========//
__global__ void yuv2rgb1(unsigned char *d_y ,unsigned char *d_u,unsigned char *d_v,unsigned char *d_r,unsigned char *d_g,unsigned char *d_b)
{
	__shared__ unsigned char ydata[cysize]; //30720
	__shared__ unsigned char udata[cuvsize]; //7680
	__shared__ unsigned char vdata[cuvsize]; //7680 (yuv = 46080)
	float rr=0,gg=0,bb=0;
	int ind_uv=0,ind=0;
	ind=blockDim.x*blockIdx.x+threadIdx.x; //640*i(0~47)+j(0~639)
	ind_uv=((blockDim.x)>>1)*(blockIdx.x>>1)+(threadIdx.x>>1);
	for(int i=0;i<cui;i++)
	{
		// get the yuv data from global memory to shared memory
		ydata[ind]=d_y[ind+(i*cysize)];
		udata[ind_uv]=d_u[ind_uv+(i*cuvsize)];
		vdata[ind_uv]=d_v[ind_uv+(i*cuvsize)];
		//===================================================			
		rr=ydata[ind]+(1.13983 * (vdata[ind_uv] - 128));
        	d_r[ind+(i*cysize)]=(rr>255 ? 255:(rr<0 ? 0:rr));	
		gg= ydata[ind] - (0.39465 * (udata[ind_uv] - 128) + (0.58060 * (vdata[ind_uv] - 128)));
        	d_g[ind+(i*cysize)]=(gg>255 ? 255:(gg<0 ? 0:gg));
        	bb= ydata[ind] + (2.03211 * (udata[ind_uv] - 128));
        	d_b[ind+(i*cysize)]=(bb>255 ? 255:(bb<0 ? 0:bb));
	}	
}
//=================================//
FILE *fp=fopen("input_video/1.yuv","rb");
unsigned char *in_y=NULL,*in_u=NULL,*in_v=NULL,*out_r=NULL,*out_g=NULL,*out_b=NULL;
unsigned char *d_y,*d_r,*d_u,*d_v,*d_g,*d_b;
int main()
{
	hipEvent_t dstart,dend;
	int process_frmaecount=0;
	float during_time=0,during[140]={0},total=0;
	hipEventCreate(&dstart);
	hipEventCreate(&dend);
	//=================CPU (host memory allocte)=================//	
	in_y=(unsigned char*)malloc(sizeof(unsigned char)*ysize);
	in_u=(unsigned char*)malloc(sizeof(unsigned char)*ysize/4);
	in_v=(unsigned char*)malloc(sizeof(unsigned char)*ysize/4);
	out_r=(unsigned char*)malloc(sizeof(unsigned char)*ysize);
	out_g=(unsigned char*)malloc(sizeof(unsigned char)*ysize);
	out_b=(unsigned char*)malloc(sizeof(unsigned char)*ysize);
	//==========================================================//
	//===============GPU (device memory allocte)================//
	hipMalloc((void **)&d_y,sizeof(unsigned char)*ysize);
	hipMalloc((void **)&d_u,sizeof(unsigned char)*ysize/4);
	hipMalloc((void **)&d_v,sizeof(unsigned char)*ysize/4);
	hipMalloc((void **)&d_r,sizeof(unsigned char)*ysize);
	hipMalloc((void **)&d_g,sizeof(unsigned char)*ysize);
	hipMalloc((void **)&d_b,sizeof(unsigned char)*ysize);
	
	hipMemset(d_r,0,sizeof(unsigned char)*ysize);
	hipMemset(d_g,0,sizeof(unsigned char)*ysize);
	hipMemset(d_b,0,sizeof(unsigned char)*ysize);
	//==========================================================//
	Mat Frame(480,640,CV_8UC3);
	while(1)
	{
		if(fread(in_y,1,ysize,fp)!=NULL)
		{
		
			fread(in_u,1,ysize/4,fp);
			fread(in_v,1,ysize/4,fp);
			//============ data from CPU to GPU =============//	
			hipMemcpy(d_y,in_y,sizeof(unsigned char)*ysize,hipMemcpyHostToDevice);
			hipMemcpy(d_u,in_u,sizeof(unsigned char)*ysize/4,hipMemcpyHostToDevice);
			hipMemcpy(d_v,in_v,sizeof(unsigned char)*ysize/4,hipMemcpyHostToDevice);
			//===============================================//
			hipEventRecord(dstart, 0);
			yuv2rgb1 <<<blks,thnum>>>(d_y,d_u,d_v,d_r,d_g,d_b); //yuv2rgb kernel function
			hipEventRecord(dend, 0);
			hipEventSynchronize(dend);
			hipEventElapsedTime(&during_time, dstart, dend);
			during[process_frmaecount]=during_time;
			process_frmaecount++;
			printf("frame%d using time=%fms\n",process_frmaecount,during[process_frmaecount-1]);		
			//============ data from GPU to CPU =============//
			hipMemcpy(out_r,d_r,sizeof(unsigned char)*ysize,hipMemcpyDeviceToHost);
			hipMemcpy(out_g,d_g,sizeof(unsigned char)*ysize,hipMemcpyDeviceToHost);
			hipMemcpy(out_b,d_b,sizeof(unsigned char)*ysize,hipMemcpyDeviceToHost);	
			//===============================================//
			//==========placed the bgr componet to Frame data struct=========//
			for(int i=0;i<h*w;i++)
			{
				Frame.data[3*i]=out_b[i];
				Frame.data[3*i+1]=out_g[i];
				Frame.data[3*i+2]=out_r[i];
			}
			//==============================================================//
			//=============display the result===============//
			imshow("transsmisson",Frame);
			cvWaitKey(10);
		}	
		if(cvWaitKey(1)>=0) break;	
	}
	for(int i=0;i<process_frmaecount;i++)
		total+=during[i];
	printf("GPU average per frame using time=%f ms\n",total/process_frmaecount);
	printf("finish job\n");
	free(in_y),free(in_u),free(in_v);
	free(out_r),free(out_g),free(out_b);
	hipFree(d_y),hipFree(d_u),hipFree(d_v),hipFree(d_r),hipFree(d_g),hipFree(d_b);
	fclose(fp);
	return 0;
}
