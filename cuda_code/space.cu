#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <memory.h>
/*-- grid -> block -> thread --*/
#define size 12
#define thread_ 3
#define block_ 4 
#define times thread_*block_
/*=============================*/
/*--struct index newtype--*/
struct ind
{
	int block;
	int thread;
};
typedef ind INDEX;
/*========================*/
/*--paralle kernel function--*/
__global__ void share_memory(INDEX* tem_dev)
{
	int dev_block=blockIdx.x;
	int dev_thread=threadIdx.x;
	int thread_of_block_num=blockDim.x;
	int thread_location_of_array=dev_block*thread_of_block_num+dev_thread;
	/*--threads write own block & threadIdx--*/
	tem_dev[thread_location_of_array].block=dev_block;
	tem_dev[thread_location_of_array].thread=dev_thread;
}
/*==========================*/
int main()
{	
	/*--set host & device--*/
	INDEX* host_temp=NULL;
	INDEX *device;
	hipError_t error;
	/*--set device memory & host memory--*/
	host_temp=(INDEX*)malloc(sizeof(INDEX)*size);
	error=hipMalloc((void**) &(device),sizeof(INDEX)*size);
	printf("cudaMalloc():%s\n",hipGetErrorString(error));
	/*--call device kernel--*/
	dim3 blocksize(640,480,4);
	dim3 threadnum(3,3);
	share_memory<<< blocksize, threadnum>>>(device);
	
	hipMemcpy(host_temp,device,sizeof(INDEX)*size,hipMemcpyDeviceToHost);
	
	for(int i=0;i<times;i++)
		printf("host_temp[%d]={block:%d,thread:%d}\n",i,host_temp[i].block,host_temp[i].thread);
	free(host_temp);
	hipFree(device);
	return 0;
}
