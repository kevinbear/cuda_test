#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <memory.h>
#define num 100
int* host_a = NULL;
int* host_b = NULL;

int main()
{
	bool flag=true;
	/*set host_memory*/
	host_a=(int*)malloc(sizeof(int)*num);
	host_b=(int*)malloc(sizeof(int)*num);
	memset(host_a,0,num);
	memset(host_b,0,num);

	/*initialization*/	
	for(int i=0;i<num;i++)
	{
		host_a[i]=i+1;
	}

	/*set device memory*/
	int* device;
	hipError_t error;
	error=hipMalloc((void**) &device,sizeof(int)*num);
	printf("cudaMalloc():%s\n",hipGetErrorString(error)); //set memory fail print errror

	error=hipMemcpy(device,host_a,sizeof(int)*num,hipMemcpyHostToDevice);// host_a -> device
	printf("cudaMemcpy(host_a => device:%s)\n",hipGetErrorString(error));

	error=hipMemcpy(host_b,device,sizeof(int)*num,hipMemcpyDeviceToHost);// device -> host_b
	printf("cudaMemcpy(device => host_a:%s)\n",hipGetErrorString(error));

	for(int i=0;i<num;i++)
	{
		if(host_a[i]!=host_b[i])
		{
			flag=false;
			break;
		}
	}
	printf("check host_a==host_b%s",flag?"pass":"worng");
	
	error=hipFree(device);//check cuda malloc free memory space
	printf("cudaFree(device):%s\n",hipGetErrorString(error));
	free(host_a);
	free(host_b);
	hipFree(device);
	return 0;
}
