#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define DATA_SIZE 1048576

int data[DATA_SIZE];

bool InitCUDA(){

	int count;


	hipGetDeviceCount(&count);
	if (count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}


	int i;
	for (i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if (prop.major >= 1) {
				break;
			}
		}
	}

	if (i == count) {
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}

	hipSetDevice(i);
	return true;
}

void GenerateNumbers(int *number, int size){
	for (int i = 0; i < size; i++) {
		number[i] = rand() % 10;
	}
}

__global__ static void sumOfSquares(int *num, int* result, clock_t* time) {

	int sum = 0;
	int i;
	clock_t start = clock();
	for (i = 0; i < DATA_SIZE; i++) {
		sum += num[i] * num[i];
	}
	*result = sum;
	*time = clock() - start;
}

int main() {
	if (!InitCUDA()) {
		return 0;
	}
	int nDevices;
	printf("CUDA initialized.\n");

	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++)
	{
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device %d has compute capability %d.%d.\n", i, prop.major, prop.minor);
		printf("NO.1\n");
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		//printf("  Memory Clock Rate (KHz): %d\n",
		//	prop.memoryClockRate);
		//printf("  Memory Bus Width (bits): %d\n",
		//	prop.memoryBusWidth);
		//printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
		//	2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);
		//printf("Maximum number of threads per block %d\n", prop.maxThreadsPerBlock);
		printf(" clockRate%d %f\n", prop.clockRate, ((float)prop.clockRate)/1000000);
		printf(" totalGlobalMem    %lu\n", prop.totalGlobalMem);
		printf(" maxThreadsPerBlock  %d\n", prop.maxThreadsPerBlock);
	}
	GenerateNumbers(data, DATA_SIZE);

	int* gpudata, *result;
	clock_t* time;

	hipMalloc((void**)&gpudata, sizeof(int)* DATA_SIZE);
	hipMalloc((void**)&result, sizeof(int));
	hipMalloc((void**)&time, sizeof(clock_t));


	hipMemcpy(gpudata, data, sizeof(int)* DATA_SIZE, hipMemcpyHostToDevice);

	int sum;
	clock_t time_used;
	clock_t realtime=clock();
	sumOfSquares << <1, 1, 0 >> >(gpudata, result, time);


	hipMemcpy(&sum, result, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&time_used, time, sizeof(clock_t), hipMemcpyDeviceToHost);
	hipFree(gpudata);
	hipFree(result);
	time_used = clock() - realtime;






	clock_t cpuclock = clock();
	int sum1 = 0;
	for (int i = 0; i < DATA_SIZE; i++) {
		sum1 += data[i] * data[i];
	}	
	printf("NO.2\n");
	printf("cuda sum: %d, CPU sum: %d\n", sum,sum1);
	clock_t cpuend = clock();
	//printf("CPU sum: %d\n", sum);
	printf("NO.3\n");
	printf("GPUtime %f\n", (float)time_used / CLOCKS_PER_SEC);
	printf("clocktime %f\n", (float)(cpuend - cpuclock) / CLOCKS_PER_SEC);
	system("PAUSE");
	return 0;
}
// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	//addKernel << <1, size >> >(dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
